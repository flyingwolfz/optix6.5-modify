#include "hip/hip_runtime.h"

#include "tutorial.h"
#include <optixu/optixu_aabb.h>
#include <optix_world.h>

#include "helpers.h"

struct PerRayData_radiance
{
	float3 result;
	float  importance;
	int    depth;
};

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
//
// 
// Dielectric surface shader
//
rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(float, fresnel_exponent, , );
rtDeclareVariable(float, fresnel_minimum, , );
rtDeclareVariable(float, fresnel_maximum, , );
rtDeclareVariable(float, refraction_index, , );
rtDeclareVariable(int, refraction_maxdepth, , );
rtDeclareVariable(int, reflection_maxdepth, , );
rtDeclareVariable(float3, refraction_color, , );
rtDeclareVariable(float3, reflection_color, , );
rtDeclareVariable(float3, extinction_constant, , );
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(int, max_depth, , );
RT_PROGRAM void glass_closest_hit_radiance()
{
	// intersection vectors
	const float3 h = ray.origin + t_hit * ray.direction;            // hitpoint
	const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
	const float3 i = ray.direction;                                            // incident direction

	float reflection = 1.0f;
	float3 result = make_float3(0.0f);

	float3 beer_attenuation;
	if (dot(n, ray.direction) > 0) {
		// Beer's law attenuation
		beer_attenuation = exp(extinction_constant * t_hit);
	}
	else {
		beer_attenuation = make_float3(1);
	}

	// refraction
	if (prd_radiance.depth < min(refraction_maxdepth, max_depth))
	{
		float3 t;                                                            // transmission direction
		if (refract(t, i, n, refraction_index))
		{

			// check for external or internal reflection
			float cos_theta = dot(i, n);
			if (cos_theta < 0.0f)
				cos_theta = -cos_theta;
			else
				cos_theta = dot(t, n);

			reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

			float importance = prd_radiance.importance * (1.0f - reflection) * optix::luminance(refraction_color * beer_attenuation);
			if (importance > importance_cutoff) {
				optix::Ray ray(h, t, RADIANCE_RAY_TYPE, scene_epsilon);
				PerRayData_radiance refr_prd;
				refr_prd.depth = prd_radiance.depth + 1;
				refr_prd.importance = importance;

				rtTrace(top_object, ray, refr_prd);
				result += (1.0f - reflection) * refraction_color * refr_prd.result;
			}
			else {
				result += (1.0f - reflection) * refraction_color * cutoff_color;
			}
		}
		// else TIR
	}

	// reflection
	if (prd_radiance.depth < min(reflection_maxdepth, max_depth))
	{
		float3 r = reflect(i, n);

		float importance = prd_radiance.importance * reflection * optix::luminance(reflection_color * beer_attenuation);
		if (importance > importance_cutoff) {
			optix::Ray ray(h, r, RADIANCE_RAY_TYPE, scene_epsilon);
			PerRayData_radiance refl_prd;
			refl_prd.depth = prd_radiance.depth + 1;
			refl_prd.importance = importance;

			rtTrace(top_object, ray, refl_prd);
			result += reflection * reflection_color * refl_prd.result;
		}
		else {
			result += reflection * reflection_color * cutoff_color;
		}
	}

	result = result * beer_attenuation;

	prd_radiance.result = result;
}


//
// (NEW)
// Attenuates shadow rays for shadowing transparent objects
//

rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void glass_any_hit_shadow()
{
	float3 world_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float nDi = fabs(dot(world_normal, ray.direction));

	prd_shadow.attenuation *= 1 - fresnel_schlick(nDi, 5, 1 - shadow_attenuation, make_float3(1));

	rtIgnoreIntersection();
}
